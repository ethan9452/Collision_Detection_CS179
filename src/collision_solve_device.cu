#include "hip/hip_runtime.h"

#include "collision_solve_device.cuh"
// #include "project_typedefs.hpp"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <time.h> 
#include <iostream>
#include <climits>
#include <algorithm>    // std::sort
#include <math.h>
#include <vector>

#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#define BLOCKS              512
#define THREADS_PER_BLOCK   512

#define PRINT_TIMES         0

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

bool particles_colliding_g(Particle p1, Particle p2) {
    // 
    #if DEBUG == 1
    if(isnan(p1.x)) {
        cout << "Nan in particles_colliding()\n";
    }
    if(isnan(p2.x)) {
        cout << "Nan in particles_colliding()\n";
    }
    if(isnan(p1.y)) {
        cout << "Nan in particles_colliding()\n";
    }
    if(isnan(p2.y)) {
        cout << "Nan in particles_colliding()\n";
    }
    if(isnan(p1.radius)) {
        cout << "Nan in particles_colliding()\n";
    }
    if(isnan(p2.radius)) {
        cout << "Nan in particles_colliding()\n";
    }
    #endif


    return ((p1.x - p2.x) * (p1.x - p2.x)) + ((p1.y - p2.y) * (p1.y - p2.y)) <
            (p1.radius + p2.radius) * (p1.radius + p2.radius);
}




__global__
void detect_collisions_GPU_naive_kernel (   Particle * dev_particles,
                                            bool * dev_output_collisions,
                                            unsigned int num_particles ) {

    unsigned pidx1 = blockIdx.x * blockDim.x + threadIdx.x;

    Particle * p1;
    Particle * p2;
    bool col;

    

    while(pidx1 < num_particles) {

        for(unsigned int pidx2 = pidx1 + 1; pidx2 < num_particles; pidx2++) {

            p1 = dev_particles + pidx1;
            p2 = dev_particles + pidx2; 

            col = ((p1->x - p2->x) * (p1->x - p2->x)) + ((p1->y - p2->y) * (p1->y - p2->y)) <
                        (p1->radius + p2->radius) * (p1->radius + p2->radius);      

            if(col == true) {
                dev_output_collisions[(pidx1 * num_particles) + pidx2] = true;
                dev_output_collisions[(pidx2 * num_particles) + pidx1] = true;
            }
        }

        pidx1 += blockDim.x * gridDim.x;
    }

}


/*
Implentation of naive algorithm on GPU.


Index (x, y) is indexed by arr[(x * num_particles) + y]

*/
void detect_collisions_GPU_naive(Particle * particles, 
                                bool * output_collisions, 
                                unsigned int num_particles, 
                                float * comp_time, 
                                Particle * dev_particles,
                                bool * dev_output_collisions ) {


    const clock_t begin_time = clock();


#if PRINT_TIMES == 1
    const clock_t copy_start_time = clock(); 
#endif

    // Set values for data on GPU
    gpuErrchk(hipMemset(dev_output_collisions, (int)false, sizeof(bool) * num_particles * num_particles));
    

#if PRINT_TIMES == 1
    hipDeviceSynchronize();
    float cpoy_runtime = float( clock () - copy_start_time ) /  CLOCKS_PER_SEC;  
    printf("First copy takes %fs\n", cpoy_runtime);
#endif



#if PRINT_TIMES == 1
    const clock_t kernel_start_time = clock();
#endif
    
    // Call kernel
    detect_collisions_GPU_naive_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_particles, dev_output_collisions, num_particles);
    
#if PRINT_TIMES == 1
    hipDeviceSynchronize();
    float kernel_runtime = float( clock () - kernel_start_time ) /  CLOCKS_PER_SEC;  
    printf("Kernel takes %fs\n", kernel_runtime);
#endif

    // check for an error in kernal call
    hipError_t err2 = hipGetLastError();
    if(err2 != hipSuccess) {
        printf("%s\n", hipGetErrorString(err2));
    }

#if PRINT_TIMES == 1
    const clock_t cpoy_back_start_time = clock(); 
#endif    

    // Copy data back
    gpuErrchk(hipMemcpy(output_collisions, dev_output_collisions, sizeof(bool) * num_particles * num_particles, hipMemcpyDeviceToHost));
    
#if PRINT_TIMES == 1
    hipDeviceSynchronize(); 
    float copy_back_runtime = float( clock () - cpoy_back_start_time ) /  CLOCKS_PER_SEC;  
    printf("Copy back takes %fs\n", copy_back_runtime);
#endif

    *comp_time = float( clock () - begin_time ) /  CLOCKS_PER_SEC;
}


__global__
void detect_collisions_GPU_optimized1_kernal_s1(  unsigned int num_particles, 
                                                    Particle * dev_particles,       // len: num_particles
                                                    bool * dev_output_collisions,   // len: num_particles^2. Index (x, y) is indexed by arr[(x * num_particles) + y]
                                                    ParticleBound * bounds_x,       // len: num_particles * 2
                                                    ParticleBound * bounds_y,       // len: num_particles * 2
                                                    int * dev_active_particles,     // len: num_particles
                                                    int * dev_active_particles_len, // a single int
                                                    float * x_sum,
                                                    float * y_sum)
{

    extern __shared__ float locations_for_sum[]; // first 'THREADS_PER_BLOCK' are x, second 'THREADS_PER_BLOCK' are y
                                                 // we do redection to find sum (and therfore mean) of x and y positions
    locations_for_sum[threadIdx.x] = 0.; // set x to 0
    locations_for_sum[threadIdx.x + THREADS_PER_BLOCK] = 0.; // set y to 0

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Fill bounds_x and bound_y
    while(i < num_particles) {

        bounds_x[2 * i].position = dev_particles[i].x - dev_particles[i].radius;
        bounds_x[2 * i].is_begin = true;
        bounds_x[2 * i].particle_idx = i;

        bounds_x[2 * i + 1].position = dev_particles[i].x + dev_particles[i].radius;
        bounds_x[2 * i + 1].is_begin = false;
        bounds_x[2 * i + 1].particle_idx = i;

        locations_for_sum[threadIdx.x] = dev_particles[i].x;

        bounds_y[2 * i].position = dev_particles[i].y - dev_particles[i].radius;
        bounds_y[2 * i].is_begin = true;
        bounds_y[2 * i].particle_idx = i;

        bounds_y[2 * i + 1].position = dev_particles[i].y + dev_particles[i].radius;
        bounds_y[2 * i + 1].is_begin = false;
        bounds_y[2 * i + 1].particle_idx = i;

        locations_for_sum[threadIdx.x + THREADS_PER_BLOCK] = dev_particles[i].y;


        __syncthreads();
        // Implement reduction to get sums
        for(uint s = blockDim.x / 2; s > 0; s>>=1) {
            if(threadIdx.x < s) {
                locations_for_sum[threadIdx.x] += locations_for_sum[threadIdx.x + s];
                locations_for_sum[threadIdx.x + THREADS_PER_BLOCK] += locations_for_sum[threadIdx.x + THREADS_PER_BLOCK + s];
            }
            __syncthreads();
        }

        if(threadIdx.x == 0) {
            atomicAdd(x_sum, locations_for_sum[0]);
            atomicAdd(y_sum, locations_for_sum[THREADS_PER_BLOCK]);
        }
        __syncthreads();

        i += blockDim.x * gridDim.x;
    }

}

__global__
void detect_collisions_GPU_optimized1_kernal_s2(  unsigned int num_particles, 
                                                    Particle * dev_particles,       // len: num_particles
                                                    bool * dev_output_collisions,   // len: num_particles^2. Index (x, y) is indexed by arr[(x * num_particles) + y]
                                                    ParticleBound * bounds_x,       // len: num_particles * 2
                                                    ParticleBound * bounds_y,       // len: num_particles * 2
                                                    int * dev_active_particles,     // len: num_particles
                                                    int * dev_active_particles_len, // a single int
                                                    float * x_mean,
                                                    float * y_mean,
                                                    float * x_var,
                                                    float * y_var )
{

    extern __shared__ float vars_shared[]; // The first 'THREADS_PER_BLOCK' elements are for x, second are for y

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Turn sum into mean
    if(i == 0) {
        *x_mean = *x_mean / (float) num_particles;
        *y_mean = *y_mean / (float) num_particles;
    }
    __syncthreads();


    // Find variance
    while(i < num_particles * 2) {

        vars_shared[threadIdx.x] = (bounds_x[i].position - *x_mean) * (bounds_x[i].position - *x_mean); // x axis
        vars_shared[threadIdx.x + THREADS_PER_BLOCK] = (bounds_y[i].position - *y_mean) * (bounds_y[i].position - *y_mean); // y axis

        __syncthreads();

        // Reduction
        for(uint s = blockDim.x / 2; s > 0; s>>=1) {
            if(threadIdx.x < s) {
                vars_shared[threadIdx.x] += vars_shared[threadIdx.x + s];
                vars_shared[threadIdx.x + THREADS_PER_BLOCK] += vars_shared[threadIdx.x + THREADS_PER_BLOCK + s];
            }
            __syncthreads();
        }

        if(threadIdx.x == 0) {
            atomicAdd(x_var, vars_shared[0]);
            atomicAdd(y_var, vars_shared[THREADS_PER_BLOCK]);
        }
        __syncthreads();


        i += blockDim.x * gridDim.x;
    }

}

void detect_collisions_GPU_optimized1(Particle * particles, 
                                    bool * output_collisions, 
                                    unsigned int num_particles, 
                                    float * comp_time,
                                    Particle * dev_particles,
                                    bool * dev_output_collisions,
                                    ParticleBound * bounds_x,
                                    ParticleBound * bounds_y,
                                    int * dev_active_particles, // indexes (wrt 'particles') of the active particles
                                    int * dev_active_particles_len ) {

    // Not including malloc in the time measurement bc in practice, we would only malloc once. Everything we are 
    // tracking time for is something that would be executed in a loop
    float * dev_x_mean; 
    float * dev_y_mean;
    gpuErrchk(hipMalloc((void**)&dev_x_mean, sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_y_mean, sizeof(float)));

    gpuErrchk(hipMemset(dev_x_mean, 0, sizeof(float)));
    gpuErrchk(hipMemset(dev_y_mean, 0, sizeof(float)));

    float * dev_x_var; 
    float * dev_y_var;
    gpuErrchk(hipMalloc((void**)&dev_x_var, sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_y_var, sizeof(float)));

    gpuErrchk(hipMemset(dev_x_var, 0, sizeof(float)));
    gpuErrchk(hipMemset(dev_y_var, 0, sizeof(float)));

    const clock_t begin_time = clock();


    // Set initial values for data on GPU
    gpuErrchk(hipMemset(dev_output_collisions, (int)false, sizeof(bool) * num_particles * num_particles));


    // Call kernel to set bounds arrays and calculate sums (of x and y positions) for means.
    detect_collisions_GPU_optimized1_kernal_s1<<<BLOCKS, THREADS_PER_BLOCK, sizeof(float) * THREADS_PER_BLOCK * 2>>>(num_particles, 
        dev_particles, dev_output_collisions, bounds_x, bounds_y, dev_active_particles, dev_active_particles_len, dev_x_mean, dev_y_mean);

    // check for an error in kernal call
    hipError_t err1 = hipGetLastError();
    if(err1 != hipSuccess) {
        printf("%s\n", hipGetErrorString(err1));
    }


    /// Call kernel to find variance in x and y positions 
    detect_collisions_GPU_optimized1_kernal_s2<<<BLOCKS, THREADS_PER_BLOCK, sizeof(float) * THREADS_PER_BLOCK * 2>>>(num_particles, 
        dev_particles, dev_output_collisions, bounds_x, bounds_y, dev_active_particles, dev_active_particles_len, dev_x_mean, dev_y_mean, dev_x_var, dev_y_var);

    // check for an error in kernal call
    hipError_t err2 = hipGetLastError();
    if(err2 != hipSuccess) {
        printf("%s\n", hipGetErrorString(err2));
    }

    float x_var;
    float y_var;
    gpuErrchk(hipMemcpy(&x_var, dev_x_var, sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&y_var, dev_y_var, sizeof(float), hipMemcpyDeviceToHost));


    // Choose higher variance
    ParticleBound * chosen_bounds; // Device
    if(x_var > y_var) {
        chosen_bounds = bounds_x;
    }
    else {
        chosen_bounds = bounds_y;
    }


    // Sort the chosen_bounds array
    // For this, I used the Cuda Thrust library, which has a parallel sorting function.
    thrust::sort(thrust::device, chosen_bounds, chosen_bounds + num_particles * 2);

    // check for an error in kernal call
    hipError_t err3 = hipGetLastError();
    if(err3 != hipSuccess) {
        printf("%s\n", hipGetErrorString(err3));
    }

    ParticleBound * bounds_chosen = new ParticleBound[2*num_particles]; // Host
    hipMemcpy(bounds_chosen, chosen_bounds, sizeof(ParticleBound) * num_particles * 2, hipMemcpyDeviceToHost);

    // List of particle_idx's that are 'active', ie: we are still inside its bounds
    std::vector<unsigned int> active_particles; 

    // Iterate through the sorted list of bounds
    // Note: did not make this into a kernel for reasons discussed in the readme
    for(unsigned long int i = 0; i < 2*num_particles; i++) {
        if(bounds_chosen[i].is_begin == true) {
            // Elements of active_particles are all potential collisions.
            // Check with each element of active_particles
            for(unsigned int active_idx = 0; active_idx < active_particles.size(); active_idx++) {
                
                if(particles_colliding_g(particles[bounds_chosen[i].particle_idx], particles[active_particles[active_idx]])) {

                    output_collisions[(bounds_chosen[i].particle_idx * num_particles) + active_particles[active_idx]] = true;
                    output_collisions[(active_particles[active_idx] * num_particles) + bounds_chosen[i].particle_idx] = true;
                }
            }
            active_particles.push_back(bounds_chosen[i].particle_idx);

        }
        else {
            // Remove bounds_chosen[i].particle_idx from active_particles
            active_particles.erase(remove(active_particles.begin(), active_particles.end(), bounds_chosen[i].particle_idx), active_particles.end());
        }
        
    }


    // Copy data back to host
    // gpuErrchk(hipMemcpy(output_collisions, dev_output_collisions, sizeof(bool) * num_particles * num_particles, hipMemcpyDeviceToHost));


    *comp_time = float( clock () - begin_time ) /  CLOCKS_PER_SEC;

    hipFree(dev_x_mean);
    hipFree(dev_y_mean);
    hipFree(dev_x_var);
    hipFree(dev_y_var);

}

// void detect_collisions_GPU_optimized2(Particle * particles, 
//                                     bool * output_collisions, 
//                                     unsigned int num_particles, 
//                                     float * comp_time) {

// }




