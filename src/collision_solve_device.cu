
#include "collision_solve_device.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <time.h> 
#include <iostream>
#include <climits>
#include <algorithm>    // std::sort
#include <vector>
#include <math.h>








void detect_collisions_GPU_naive(Particle * particles, 
								bool * output_collisions, 
								unsigned int num_particles, 
								float * comp_time) {

}

void detect_collisions_GPU_optimized1(Particle * particles, 
									bool * output_collisions, 
									unsigned int num_particles, 
									float * comp_time) {

}

void detect_collisions_GPU_optimized2(Particle * particles, 
									bool * output_collisions, 
									unsigned int num_particles, 
									float * comp_time) {

}